#include "hip/hip_runtime.h"
#include "helper.h"

// 根据上一步计算出的可以复原的query_id进行状态恢复
__global__ void recover_block_system_cache(int *recover_block_list, // [bsz]
                                           int *recover_len,
                                           bool *stop_flags,
                                           int *seq_lens_this_time,
                                           int *ori_seq_lens_encoder,
                                           int *ori_seq_lens_decoder,
                                           int *seq_lens_encoder,
                                           int *seq_lens_decoder,
                                           int *block_tables,
                                           int *free_list,
                                           int *free_list_len,
                                           int64_t *input_ids,
                                           int64_t *pre_ids,
                                           int64_t *step_idx,
                                           int *encoder_block_lens,
                                           int *used_list_len,
                                           const int64_t *next_tokens,
                                           const int64_t *first_token_ids,
                                           const int bsz,
                                           const int block_num_per_seq,
                                           const int length,
                                           const int pre_id_length) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    __shared__ int ori_free_list_len;
    if (bid < recover_len[0]) {
        const int recover_id = recover_block_list[bid];
        const int ori_seq_len_encoder = ori_seq_lens_encoder[recover_id];
        const int step_idx_now = step_idx[recover_id];
        const int seq_len = ori_seq_len_encoder + step_idx_now;
        const int encoder_block_len = encoder_block_lens[recover_id];
        const int decoder_used_len = used_list_len[recover_id];
        int *block_table_now = block_tables + recover_id * block_num_per_seq;
        int64_t *input_ids_now = input_ids + recover_id * length;
        int64_t *pre_ids_now = pre_ids + recover_id * pre_id_length;
        if (tid == 0) {
            seq_lens_this_time[recover_id] = seq_len;
            seq_lens_encoder[recover_id] = seq_len;
            seq_lens_decoder[recover_id] = ori_seq_lens_decoder[recover_id];
            stop_flags[recover_id] = false;
            input_ids_now[ori_seq_len_encoder + step_idx_now - 1] = next_tokens[recover_id]; // next tokens
            input_ids_now[0] = first_token_ids[recover_id]; // set first prompt token
            const int ori_free_list_len_tid0 = atomicSub(free_list_len, decoder_used_len);
            ori_free_list_len = ori_free_list_len_tid0;
#ifdef DEBUG_STEP
            printf("seq_id: %d, ori_seq_len_encoder: %d, step_idx_now: %d, seq_len: %d, ori_free_list_len_tid0: %d, ori_free_list_len: %d\n", 
                    recover_id, ori_seq_len_encoder, step_idx_now, seq_len, ori_free_list_len_tid0, ori_free_list_len);
#endif
        }
        __syncthreads();
        // 恢复block table
        for (int i = tid; i < decoder_used_len; i += blockDim.x) {
            block_table_now[encoder_block_len + i] = free_list[ori_free_list_len - i - 1];
        }
        // 恢复input_ids
        for (int i = tid; i < step_idx_now - 1; i += blockDim.x) {
            input_ids_now[ori_seq_len_encoder + i] = pre_ids_now[i + 1];
        }
    }

    if (bid == 0 && tid == 0) {
        recover_len[0] = 0;
    }
}

void StepSystemCache(const paddle::Tensor& stop_flags,
                     const paddle::Tensor& seq_lens_this_time,
                     const paddle::Tensor& ori_seq_lens_encoder,
                     const paddle::Tensor& ori_seq_lens_decoder,
                     const paddle::Tensor& seq_lens_encoder,
                     const paddle::Tensor& seq_lens_decoder,
                     const paddle::Tensor& block_tables, // [bsz, block_num_per_seq]
                     const paddle::Tensor& encoder_block_lens,
                     const paddle::Tensor& is_block_step,
                     const paddle::Tensor& step_block_list,
                     const paddle::Tensor& step_lens,
                     const paddle::Tensor& recover_block_list,
                     const paddle::Tensor& recover_lens,
                     const paddle::Tensor& need_block_list,
                     const paddle::Tensor& need_block_len,
                     const paddle::Tensor& used_list_len, 
                     const paddle::Tensor& free_list,
                     const paddle::Tensor& free_list_len,
                     const paddle::Tensor& input_ids,
                     const paddle::Tensor& pre_ids,
                     const paddle::Tensor& step_idx,
                     const paddle::Tensor& next_tokens,
                     const paddle::Tensor& first_token_ids,
                     const int block_size,
                     const int encoder_decoder_block_num) {
    auto cu_stream = seq_lens_this_time.stream();
    const int bsz = seq_lens_this_time.shape()[0];
    const int block_num_per_seq = block_tables.shape()[1];
    const int length = input_ids.shape()[1];
    const int pre_id_length = pre_ids.shape()[1];
    constexpr int BlockSize = 256; // bsz <= 256
    const int max_decoder_block_num = length / block_size;
    // const int max_decoder_block_num = 2048 / block_size - encoder_decoder_block_num;
#ifdef DEBUG_STEP
    printf("bsz: %d, block_num_per_seq: %d, length: %d, max_decoder_block_num: %d\n", bsz, block_num_per_seq, length, max_decoder_block_num);
#endif
    free_and_dispatch_block<<<1, BlockSize, 0, cu_stream>>>(
        const_cast<bool*>(stop_flags.data<bool>()),
        const_cast<int*>(seq_lens_this_time.data<int>()),
        const_cast<int*>(seq_lens_decoder.data<int>()),
        const_cast<int*>(block_tables.data<int>()),
        const_cast<int*>(encoder_block_lens.data<int>()),
        const_cast<bool*>(is_block_step.data<bool>()),
        const_cast<int*>(step_block_list.data<int>()),
        const_cast<int*>(step_lens.data<int>()),
        const_cast<int*>(recover_block_list.data<int>()),
        const_cast<int*>(recover_lens.data<int>()),
        const_cast<int*>(need_block_list.data<int>()),
        const_cast<int*>(need_block_len.data<int>()),
        const_cast<int*>(used_list_len.data<int>()),
        const_cast<int*>(free_list.data<int>()),
        const_cast<int*>(free_list_len.data<int>()),
        const_cast<int64_t*>(first_token_ids.data<int64_t>()),
        bsz,
        block_size,
        block_num_per_seq,
        max_decoder_block_num
    );
#ifdef DEBUG_STEP
    hipDeviceSynchronize();
#endif
    auto cpu_recover_lens = recover_lens.copy_to(paddle::CPUPlace(), false);
    const int grid_size = cpu_recover_lens.data<int>()[0];
#ifdef DEBUG_STEP
    printf("grid_size2 %d\n", grid_size);
#endif
    if (grid_size > 0) {
        recover_block_system_cache<<<grid_size, BlockSize, 0, cu_stream>>>(
            const_cast<int*>(recover_block_list.data<int>()),
            const_cast<int*>(recover_lens.data<int>()),
            const_cast<bool*>(stop_flags.data<bool>()),
            const_cast<int*>(seq_lens_this_time.data<int>()),
            const_cast<int*>(ori_seq_lens_encoder.data<int>()),
            const_cast<int*>(ori_seq_lens_decoder.data<int>()),
            const_cast<int*>(seq_lens_encoder.data<int>()),
            const_cast<int*>(seq_lens_decoder.data<int>()),
            const_cast<int*>(block_tables.data<int>()),
            const_cast<int*>(free_list.data<int>()),
            const_cast<int*>(free_list_len.data<int>()),
            const_cast<int64_t*>(input_ids.data<int64_t>()),
            const_cast<int64_t*>(pre_ids.data<int64_t>()),
            const_cast<int64_t*>(step_idx.data<int64_t>()),
            const_cast<int*>(encoder_block_lens.data<int>()),
            const_cast<int*>(used_list_len.data<int>()),
            next_tokens.data<int64_t>(),
            first_token_ids.data<int64_t>(),
            bsz,
            block_num_per_seq,
            length,
            pre_id_length
        );
#ifdef DEBUG_STEP
        hipDeviceSynchronize();
#endif
    }
}

PD_BUILD_OP(step_system_cache)
    .Inputs({"stop_flags", 
             "seq_lens_this_time",
             "ori_seq_lens_encoder",
             "ori_seq_lens_decoder",
             "seq_lens_encoder",
             "seq_lens_decoder",
             "block_tables",
             "encoder_block_lens",
             "is_block_step",
             "step_block_list",
             "step_lens",
             "recover_block_list",
             "recover_lens",
             "need_block_list",
             "need_block_len",
             "used_list_len",
             "free_list",
             "free_list_len",
             "input_ids",
             "pre_ids",
             "step_idx",
             "next_tokens",
             "first_token_ids"})
    .Attrs({"block_size: int",
            "encoder_decoder_block_num: int"})
    .Outputs({"stop_flags_out",
              "seq_lens_this_time_out",
              "seq_lens_encoder_out",
              "seq_lens_decoder_out",
              "block_tables_out",
              "encoder_block_lens_out",
              "is_block_step_out",
              "step_block_list_out",
              "step_lens_out",
              "recover_block_list_out",
              "recover_lens_out",
              "need_block_list_out",
              "need_block_len_out",
              "used_list_len_out",
              "free_list_out",
              "free_list_len_out",
              "input_ids_out",
              "first_token_ids_out"})
    .SetInplaceMap({{"stop_flags", "stop_flags_out"},
                    {"seq_lens_this_time", "seq_lens_this_time_out"},
                    {"seq_lens_encoder", "seq_lens_encoder_out"},
                    {"seq_lens_decoder", "seq_lens_decoder_out"},
                    {"block_tables", "block_tables_out"},
                    {"encoder_block_lens", "encoder_block_lens_out"},
                    {"is_block_step", "is_block_step_out"},
                    {"step_block_list", "step_block_list_out"},
                    {"step_lens", "step_lens_out"},
                    {"recover_block_list", "recover_block_list_out"},
                    {"recover_lens", "recover_lens_out"},
                    {"need_block_list", "need_block_list_out"},
                    {"need_block_len", "need_block_len_out"},
                    {"used_list_len", "used_list_len_out"},
                    {"free_list", "free_list_out"},
                    {"free_list_len", "free_list_len_out"},
                    {"input_ids", "input_ids_out"},
                    {"first_token_ids", "first_token_ids_out"}})
    .SetKernelFn(PD_KERNEL(StepSystemCache));
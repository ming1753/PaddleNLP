#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"
#include "paddle/extension.h"
#include "paddle/phi/core/memory/memcpy.h"

template <int THREADBLOCK_SIZE>
__global__ void GetMaxLenKernel(const int *seq_lens,
                                const int *seq_lens_this_time,
                                const int *seq_lens_encoder,
                                const int *seq_lens_this_time_merged,
                                const int *seq_lens_encoder_merged,
                                const int *seq_mapping,
                                const int *system_lens,
                                int *max_lens,
                                const int batch_size) {
  const int tid = threadIdx.x;

  typedef hipcub::BlockReduce<int, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int max_len_this_time_this_thread = 0;
  int max_len_encoder_this_thread = 0;
  int max_len_decoder_this_thread = 0;
  int max_len_this_thread = 0;
  int max_just_dec_len_this_thread = 0;
  int max_just_dec_merged_len_this_time_this_thread = 0;
  int max_system_len_this_thread = 0;
  int max_dec_len_without_system_this_thread = 0;
  for (int i = tid; i < batch_size; i += blockDim.x) {
    const int seq_len_this_time = seq_lens_this_time[i];
    max_len_this_time_this_thread = max(seq_len_this_time,
                                        max_len_this_time_this_thread);
    max_len_encoder_this_thread = max(seq_lens_encoder[i],
                                      max_len_encoder_this_thread);
    max_len_decoder_this_thread = max(seq_lens[i], max_len_decoder_this_thread);
    if (seq_len_this_time <= 0) continue;
    const int max_just_dec_len_now = seq_lens_encoder[i] > 0 ? 0 : seq_lens[i];
    max_len_this_thread = max(seq_lens[i] + seq_len_this_time,
                              max_len_this_thread);
    max_just_dec_len_this_thread = max(max_just_dec_len_this_thread,
                                       max_just_dec_len_now);
    if (system_lens) {
      const int real_bid = seq_mapping[i];
      const int system_len_now = system_lens[real_bid];
      max_system_len_this_thread = max(max_system_len_this_thread, system_len_now);
      max_dec_len_without_system_this_thread = max(max_dec_len_without_system_this_thread,
                                                   max_just_dec_len_now - system_len_now);
    }
  }
  if (system_lens) {
    for (int i = tid; i < batch_size; i += blockDim.x) {
      const int ori_seq_len_this_time = seq_lens_this_time_merged[i];
      if (ori_seq_len_this_time <= 0) continue;
      const int max_just_dec_merged_len_this_time_now = seq_lens_encoder_merged[i] > 0 ?
                                                        0 : ori_seq_len_this_time;
      max_just_dec_merged_len_this_time_this_thread = max(max_just_dec_merged_len_this_time_this_thread,
                                                          max_just_dec_merged_len_this_time_now);
    }
  }
  int total_max_len_this_time = BlockReduce(temp_storage).Reduce(max_len_this_time_this_thread, MaxOp<int>());
  int total_max_len_encoder = BlockReduce(temp_storage).Reduce(max_len_encoder_this_thread, MaxOp<int>());
  int total_max_len_decoder = BlockReduce(temp_storage).Reduce(max_len_decoder_this_thread, MaxOp<int>());
  int total = BlockReduce(temp_storage).Reduce(max_len_this_thread, MaxOp<int>());
  int total_just_dec = BlockReduce(temp_storage).Reduce(max_just_dec_len_this_thread, MaxOp<int>());
  int total_just_dec_merged = BlockReduce(temp_storage).Reduce(max_just_dec_merged_len_this_time_this_thread, MaxOp<int>());
  int total_system_len = BlockReduce(temp_storage).Reduce(max_system_len_this_thread, MaxOp<int>());
  int total_dec_len_without_system = BlockReduce(temp_storage).Reduce(max_dec_len_without_system_this_thread, MaxOp<int>());
  if (tid == 0) {
    max_lens[0] = total_max_len_this_time;
    max_lens[1] = total_max_len_encoder;
    max_lens[2] = total_max_len_decoder;
    max_lens[3] = total;
    max_lens[4] = total_just_dec;
    max_lens[5] = total_just_dec_merged;
    max_lens[6] = total_system_len;
    max_lens[7] = total_dec_len_without_system;
  }
}

void GetMaxLen(const paddle::Tensor& seq_lens_tensor,
              const paddle::Tensor& seq_lens_this_time,
              const paddle::Tensor& seq_lens_encoder,
              paddle::Tensor& max_len_tensor,
              const int batch_size) {
  constexpr int blockSize = 1024;
  GetMaxLenKernel<blockSize><<<1, blockSize, 0, seq_lens_encoder.stream()>>>(
    seq_lens_tensor.data<int>(),
    seq_lens_this_time.data<int>(),
    seq_lens_encoder.data<int>(),
    nullptr,
    nullptr,
    nullptr,
    nullptr,
    max_len_tensor.data<int>(),
    batch_size);
}


template <typename T>
inline __device__ __host__ T div_up(T m, T n) {
  return (m + n - 1) / n;
}

__global__ void split_q_block(const int* __restrict__ seq_lens_q,
                              const int* __restrict__ seq_lens_encoder,
                              int* __restrict__ batch_ids,
                              int* __restrict__ tile_ids_per_batch,
                              int* __restrict__ num_blocks_x,
                              const int bsz,
                              const int num_rows_per_block,
                              const int group_size) {
  if (threadIdx.x == 0) {
    int gridx = 0;
    int index = 0;
    for (uint32_t bid = 0; bid < bsz; bid++) {
      int seq_len = seq_lens_q[bid];
      if (seq_lens_encoder && seq_lens_encoder[bid] > 0) {
        seq_len = 0;
      }
      const int loop_times =
          div_up(seq_len * group_size, num_rows_per_block);
      for (uint32_t tile_id = 0; tile_id < loop_times; tile_id++) {
        batch_ids[index] = bid;
        tile_ids_per_batch[index++] = tile_id;
      }
      gridx += loop_times;
    }
    *num_blocks_x = gridx;
  }
}

__global__ void split_kv_block(const int* __restrict__ seq_lens_decoder,
                               const int* __restrict__ seq_lens_encoder,
                               int* __restrict__ batch_ids,
                               int* __restrict__ tile_ids_per_batch,
                               int* __restrict__ num_blocks_x,
                               const int bsz,
                               const int pad_len,
                               const int num_row_per_block) {
  if (threadIdx.x == 0) {
    int gridx = 0;
    int index = 0;
    for (uint32_t bid = 0; bid < bsz; bid++) {
      const int start_len = seq_lens_decoder[bid];
      int seq_len = seq_lens_encoder[bid] + start_len % pad_len;
      if (seq_lens_encoder[bid] == 0) {
        seq_len = 0;
      }
      const int loop_times = div_up(seq_len, num_row_per_block);
      for (uint32_t tile_id = 0; tile_id < loop_times; tile_id++) {
        batch_ids[index] = bid;
        tile_ids_per_batch[index++] = tile_id;
      }
      gridx += loop_times;
    }
    *num_blocks_x = gridx;
  }
}

template <int THREADBLOCK_SIZE>
__global__ void get_max_len_kv_ernel(int* max_seq_lens_out,
                                  const int* seq_lens_this_time,
                                  const int* seq_lens_decoder,
                                  const int batch_size) {
  const int tid = threadIdx.x;

  
  typedef hipcub::BlockReduce<int, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int max_len_this_thread = 0;
  for (int i = tid; i < batch_size; i += blockDim.x) {
    if (seq_lens_decoder[i] == 0) continue;
    max_len_this_thread = max(seq_lens_this_time[i] + seq_lens_decoder[i], max_len_this_thread);
  }
  int total = BlockReduce(temp_storage).Reduce(max_len_this_thread, MaxOp<int>());
  if (tid == 0) {
    *max_seq_lens_out = total;
  }
}

std::vector<paddle::Tensor> GetBlockShapeAndSplitKVBlock(
    const paddle::Tensor& seq_lens_encoder,
    const paddle::Tensor& seq_lens_decoder,
    const paddle::Tensor& seq_lens_this_time,
    const paddle::Tensor& cum_offsets,
    const int group_size,
    const int block_size,
    const int decoder_step_token_num) {
  auto stream = seq_lens_this_time.stream();
  int bsz = cum_offsets.shape()[0];
  
  auto max_len_tensor = GetEmptyTensor({8}, paddle::DataType::INT32, seq_lens_encoder.place());
  GetMaxLen(
    seq_lens_decoder,
    seq_lens_this_time,
    seq_lens_encoder,
    max_len_tensor,
    bsz);

  
  // max_len_this_time, max_enc_len_this_time, max_dec_len_this_time, max_enc_dec_len_this_time,
  // max_just_dec_len_this_time, max_just_dec_merged_len_this_time, max_system_len, max_just_dec_len_without_system
  auto max_len_cpu = max_len_tensor.copy_to(paddle::CPUPlace(), false);
  auto max_len_cpu_ptr = max_len_cpu.data<int>();
  int max_len_this_time = max_len_cpu_ptr[0];
  int max_enc_len_this_time = max_len_cpu_ptr[1];
  int max_dec_len_this_time = max_len_cpu_ptr[2];
  int max_enc_dec_len_this_time = max_len_cpu_ptr[3];
  int max_just_dec_len_this_time = max_len_cpu_ptr[4];
  int max_just_dec_merged_len_this_time = max_len_cpu_ptr[5];
  int max_system_len = max_len_cpu_ptr[6];
  int max_just_dec_len_without_system = max_len_cpu_ptr[7];

  // has some dummy tensor for dynamic or pir mode
  paddle::Tensor encoder_batch_ids = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor encoder_tile_ids_per_batch = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor encoder_num_blocks_x_cpu = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace()); /*cpu*/
  paddle::Tensor kv_batch_ids = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor kv_tile_ids_per_batch = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor kv_num_blocks_x_cpu = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace()); /*cpu*/
  paddle::Tensor decoder_batch_ids = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor decoder_tile_ids_per_batch = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace());
  paddle::Tensor decoder_num_blocks_x_cpu = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace()); /*cpu*/
  paddle::Tensor max_len_kv_cpu = GetEmptyTensor({1},paddle::DataType::INT32, paddle::CPUPlace()); /*cpu*/

  const int encoder_block_shape_q = get_encoder_block_shape_q();
  const int decoder_block_shape_q = get_decoder_block_shape_q();

  // max_len
  auto max_len_kv =
      GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_decoder.place());
  get_max_len_kv_ernel<128><<<1, 128, 0, stream>>>(
    max_len_kv.data<int>(),
    seq_lens_this_time.data<int>(),
    seq_lens_decoder.data<int>(),
    bsz
  );

  max_len_kv_cpu =
      max_len_kv.copy_to(paddle::CPUPlace(), false);

  if (max_enc_len_this_time > 0) {
    const uint32_t max_tile_size_per_bs_kv = div_up(max_enc_dec_len_this_time, block_size);
    kv_batch_ids = GetEmptyTensor({bsz * max_tile_size_per_bs_kv},
                                      paddle::DataType::INT32,
                                      seq_lens_encoder.place());
    kv_tile_ids_per_batch = GetEmptyTensor({bsz * max_tile_size_per_bs_kv},
                                                paddle::DataType::INT32,
                                                seq_lens_encoder.place());
    auto kv_num_blocks_x =
        GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());

    split_kv_block<<<1, 32, 0, seq_lens_encoder.stream()>>>(
      seq_lens_decoder.data<int>(),
      // sequence_lengths->data<int>(),
      seq_lens_encoder.data<int>(),
      kv_batch_ids.data<int>(),
      kv_tile_ids_per_batch.data<int>(),
      kv_num_blocks_x.data<int>(),
      bsz,
      block_size,
      block_size
    );

    kv_num_blocks_x_cpu = kv_num_blocks_x.copy_to(paddle::CPUPlace(), false);

    const uint32_t encoder_max_tile_size_per_bs_q = div_up(
        (max_enc_dec_len_this_time * group_size), encoder_block_shape_q);
    encoder_batch_ids =
        GetEmptyTensor({bsz * encoder_max_tile_size_per_bs_q},
                      paddle::DataType::INT32,
                      seq_lens_encoder.place());
    encoder_tile_ids_per_batch =
        GetEmptyTensor({bsz * encoder_max_tile_size_per_bs_q},
                      paddle::DataType::INT32,
                      seq_lens_encoder.place());
    auto encoder_num_blocks_x =
        GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());
    split_q_block<<<1, 32, 0, stream>>>(seq_lens_encoder.data<int>(),
                                        nullptr,
                                        encoder_batch_ids.data<int>(),
                                        encoder_tile_ids_per_batch.data<int>(),
                                        encoder_num_blocks_x.data<int>(),
                                        bsz,
                                        encoder_block_shape_q,
                                        group_size);
    encoder_num_blocks_x_cpu =
        encoder_num_blocks_x.copy_to(paddle::CPUPlace(), false);
  }
  if (max_just_dec_len_this_time > 0) {
    const uint32_t decoder_max_tile_size_per_bs_q =
        div_up((decoder_step_token_num * group_size), decoder_block_shape_q);

    decoder_batch_ids =
        GetEmptyTensor({bsz * decoder_max_tile_size_per_bs_q},
                      paddle::DataType::INT32,
                      seq_lens_encoder.place());
    decoder_tile_ids_per_batch =
        GetEmptyTensor({bsz * decoder_max_tile_size_per_bs_q},
                      paddle::DataType::INT32,
                      seq_lens_encoder.place());
    auto decoder_num_blocks_x =
        GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());
    split_q_block<<<1, 32, 0, stream>>>(seq_lens_this_time.data<int>(),
                                        seq_lens_encoder.data<int>(),
                                        decoder_batch_ids.data<int>(),
                                        decoder_tile_ids_per_batch.data<int>(),
                                        decoder_num_blocks_x.data<int>(),
                                        bsz,
                                        decoder_block_shape_q,
                                        group_size);
    decoder_num_blocks_x_cpu =
        decoder_num_blocks_x.copy_to(paddle::CPUPlace(), false);
  }
  return {encoder_batch_ids,
          encoder_tile_ids_per_batch,
          encoder_num_blocks_x_cpu, /*cpu*/
          kv_batch_ids,
          kv_tile_ids_per_batch,
          kv_num_blocks_x_cpu, /*cpu*/
          decoder_batch_ids,
          decoder_tile_ids_per_batch,
          decoder_num_blocks_x_cpu, /*cpu*/
          max_len_kv_cpu, /*cpu*/
          max_len_cpu};
}

std::vector<paddle::DataType> GetBlockShapeAndSplitKVBlockInferDtype(
    const paddle::DataType& seq_lens_encoder_dtype,
    const paddle::DataType& seq_lens_decoder_dtype,
    const paddle::DataType& seq_lens_this_time_dtype,
    const paddle::DataType& cum_offsets_dtype) {
  return {paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32};
}

std::vector<std::vector<int64_t>> GetBlockShapeAndSplitKVBlockInferShape(
    const std::vector<int64_t>& seq_lens_encoder_shape,
    const std::vector<int64_t>& seq_lens_decoder_shape,
    const std::vector<int64_t>& seq_lens_this_time_shape,
    const std::vector<int64_t>& cum_offsets_shape) {
  std::vector<int64_t> dynamic_shape = {-1};

  return {dynamic_shape,
          dynamic_shape,
          {1},
          dynamic_shape,
          dynamic_shape,
          {1},
          dynamic_shape,
          dynamic_shape,
          {1},
          {1},
          {8}};
}

PD_BUILD_OP(get_block_shape_and_split_kv_block)
    .Inputs({"seq_lens_encoder",
             "seq_lens_decoder",
             "seq_lens_this_time",
             "cum_offsets"})
    .Outputs({"encoder_batch_ids",
              "encoder_tile_ids_per_batch",
              "encoder_num_blocks",
              "kv_batch_ids",
              "kv_tile_ids_per_batch",
              "kv_num_blocks",
              "decoder_batch_ids",
              "decoder_tile_ids_per_batch",
              "decoder_num_blocks",
              "max_len_kv",
              "set_max_lengths"})
    .Attrs({"group_size: int",
            "block_size: int",
            "decoder_step_token_num: int"})
    .SetKernelFn(PD_KERNEL(GetBlockShapeAndSplitKVBlock))
    .SetInferShapeFn(PD_INFER_SHAPE(GetBlockShapeAndSplitKVBlockInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetBlockShapeAndSplitKVBlockInferDtype));
